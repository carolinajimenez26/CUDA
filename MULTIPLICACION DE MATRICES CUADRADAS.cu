//MULTIPLICACIÓN DE MATRICES(APLANADAS)CUADRADAS EN C++ y CUDA con tiempo
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 


__global__ void MultiplicaMatricesCU(int* A,int* B,int* C,int ancho){
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if((row<ancho)&&(col<ancho)){
		int suma=0;
		for(int k=0;k<ancho;k++){//Se mueve entre las filas de B 
			suma=suma+A[(row*ancho)+k]*B[(k*ancho)+col];
		}
		C[(row*ancho)+col]=suma;
	}	
}

__host__ void multiplicaMatrices(int* X,int filX,int colX,int* Y,int filY,int colY,int* Z){
	for(int i=0;i<filX;i++){
		for(int j=0;j<colY;j++){
			int suma=0;
			for(int k=0;k<filY;k++){
				suma=suma+X[(i*colX)+k]*Y[(k*filY)+j];
			}
			Z[(i*colY)+j]=suma;
		}	
	}
}

__host__ void imprime(int* A,int filas, int columnas){//imprime como si fuera una matriz
	for(int i = 0; i < filas; i++){
        	for(int j = 0; j < columnas; j++){
            		cout<<A[(i*columnas)+j]<<" ";
        	}
        cout<<endl;
    }
}	

__host__ void inicializa(int *A,int filas, int columnas){//inicializa arreglos
	for(int i=0;i<filas*columnas;i++){
		A[i]=1;
	}
}

int main(void){

	clock_t startCPU,endCPU,startGPU,endGPU;  
	int *A,*B,*C; //A[filA][colA],B[filB][colB],C[filA][colB]
	int *d_A,*d_B,*d_C,*h_C;
	int filA=1024,colA=1024,filB=1024,colB=1024;
	
	//-------------------------------CPU--------------------------------------------------------------------
	startCPU = clock();	

	A=(int*)malloc(filA*colA*sizeof(int)); 
	B=(int*)malloc(filB*colB*sizeof(int));
	C=(int*)malloc(filA*colB*sizeof(int));

	inicializa(A,filA,colA);
	inicializa(B,filB,colB);
	
	if(colA==filB){//para que sean multiplicables
		multiplicaMatrices(A,filA,colA,B,filB,colB,C);
		//imprime(C,filA,colB);
	}else{
		cout<<"Error, no se pueden multiplicar"<<endl;
		return 0;
	}
	
	endCPU = clock();

	double time_CPU=((double)(endCPU-startCPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la CPU fue: "<<time_CPU<<endl;
	//-------------------------------GPU--------------------------------------------------------------------
	h_C=(int*)malloc(filA*colB*sizeof(int));

	startGPU = clock();

	hipMalloc((void**)&d_A,filA*colA*sizeof(int));
	hipMalloc((void**)&d_B,filB*colB*sizeof(int));
	hipMalloc((void**)&d_C,filA*colB*sizeof(int));	
	
	//Depende directamente de la dimensión de las matrices
	dim3 dimblock(32,32,1);
	dim3 dimGrid(ceil(filA/32),ceil(colB/32),1);
	
	MultiplicaMatricesCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,1024);

	hipDeviceSynchronize();

	hipMemcpy(h_C,d_C,filA*colB*sizeof(int),hipMemcpyDeviceToHost);
	
	endGPU = clock();

	//imprime(h_C,filA,colB);
	double time_GPU=((double)(endGPU-startGPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la GPU fue: "<<time_GPU<<endl;
	//-----------------------------------------------------------------------------------
	cout<<"El tiempo de aceleramiento fue: "<<time_CPU/time_GPU<<endl;
	free(A);free(B);free(C);free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}

