//MULTIPLICACIÓN DE MATRICES(APLANADAS) EN C++ y CUDA con tiempo
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 


__global__ void MultiplicaMatricesCU(int* A,int* B,int* C,int filA,int colA,int filB,int colB){
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	// ancho=n (columnas, o sea en x)
	if((row<filA)&&(col<colB)){//C[filA][colB]
		int suma=0;
		for(int k=0;k<filB;k++){//Se mueve entre las filas de B 
			suma=suma+A[(row*colA)+k]*B[(row*filB)+k];
		}
		C[(row*colB)+col]=suma;
	}	
}

__host__ void multiplicaMatrices(int* X,int filX,int colX,int* Y,int filY,int colY,int* Z){
	for(int i=0;i<filX;i++){
		for(int j=0;j<colY;j++){
			int suma=0;
			for(int k=0;k<filY;k++){
				suma=suma+X[(i*colX)+k]*Y[(k*filY)+j];
			}
			Z[(i*colY)+j]=suma;
		}	
	}
}

__host__ void imprime(int* A,int filas, int columnas){//imprime como si fuera una matriz
	for(int i = 0; i < filas; i++){
        	for(int j = 0; j < columnas; j++){
            		cout<<A[(i*columnas)+j]<<" ";
        	}
        cout<<endl;
    }
}	

__host__ void inicializa(int *A,int filas, int columnas){//inicializa arreglos
	for(int i=0;i<filas*columnas;i++){
		A[i]=1;
	}
}

int main(void){

	clock_t startCPU,endCPU,startGPU,endGPU;  
	int *A,*B,*C; //A[filA][colA],B[filB][colB],C[filA][colB]
	int *d_A,*d_B,*d_C,*h_C;
	int filA=2,colA=2,filB=2,colB=4;
	
	//-------------------------------CPU--------------------------------------------------------------------
	startCPU = clock();	

	A=(int*)malloc(filA*colA*sizeof(int)); 
	B=(int*)malloc(filB*colB*sizeof(int));
	C=(int*)malloc(filA*colB*sizeof(int));

	inicializa(A,filA,colA);
	inicializa(B,filB,colB);
	
	if(colA==filB){//para que sean multiplicables
		multiplicaMatrices(A,filA,colA,B,filB,colB,C);
		imprime(C,filA,colB);
	}else{
		cout<<"Error, no se pueden multiplicar"<<endl;
	}
	
	endCPU = clock();

	double time_CPU=((double)(endCPU-startCPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la CPU fue: "<<time_CPU<<endl;
	//-------------------------------GPU--------------------------------------------------------------------
	h_C=(int*)malloc(filA*colB*sizeof(int));

	startGPU = clock();

	hipMalloc((void**)&d_A,filA*colA*sizeof(int));
	hipMalloc((void**)&d_B,filB*colB*sizeof(int));
	hipMalloc((void**)&d_C,filA*colB*sizeof(int));	
	
	//Depende directamente de la dimensión de las matrices
	dim3 dimblock(3,3,1);
	dim3 dimGrid(1,1,1);
	
	MultiplicaMatricesCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,filA,colA,filB,colB);

	hipDeviceSynchronize();

	hipMemcpy(h_C,d_C,filA*colB*sizeof(int),hipMemcpyDeviceToHost);
	
	endGPU = clock();

	imprime(h_C,filA,colB);
	double time_GPU=((double)(endGPU-startGPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la GPU fue: "<<time_GPU<<endl;
	//-----------------------------------------------------------------------------------
	cout<<"El tiempo de aceleramiento fue: "<<time_CPU/time_GPU<<endl;
	free(A);free(B);free(C);free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}

