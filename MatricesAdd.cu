#include "hip/hip_runtime.h"
#include <malloc.h>
#include <hip/hip_runtime.h>

#define m 10 //fila
#define n 10 //columna, matriz mxn
#define SIZE m*n //dimension de la matriz


__global__ void VectorAdd(int *a, int *b, int *c)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if((row<m)&&(col<n))
		c[i]=a[i]+b[i];
}


int main(void){

	//clock_t start = clock();

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	
	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));
	
	hipMalloc(&d_a, SIZE*sizeof(int));
	hipMalloc(&d_b, SIZE*sizeof(int));
	hipMalloc(&d_c, SIZE*sizeof(int));

	for(int i=0;i<SIZE;i++){
		a[i]=i;
		b[i]=i;
		c[i]=0;	
	}

	hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice);

	
	dim3 dimblock(10,10,1);
	dim3 dimGrid(1,1,1);
	
	VectorAdd<<<dimblock,dimGrid>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<10; i++)
		printf("%d ",c[i]);
		
	free(a);
	free(b);
	free(c);
	
	hipFree(d_a);	
	hipFree(d_b);
	hipFree(d_c);
	
	//printf("Tiempo transcurrido: %f \n ",((double)clock() - start) / CLOCKS_PER_SEC);

	return 0;
}
