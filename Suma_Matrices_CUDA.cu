//PROGRAMA QUE SUMA DOS MATRICES EN C++
#include<iostream>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 


__global__ void SumaMatricesCU(int** A,int** B,int** C,int x,int y){
	int idx=blockIdx.x*blockDim.x + threadIdx.x;//columnas
	int idy=blockIdx.y*blockDim.y + threadIdx.y;//filas
	if((idx<x)&&(idy<y)){
		C[idx][idy]=A[idx][idy]+B[idx][idy];
	}	
}

void imprimeMatriz(int** A, int x, int y){
	for(int i=0;i<x;i++){
		for(int j=0;j<y;j++){
			cout<<A[i][j]<<" ";
		}
		cout<<endl;
	}

}

void inicializaMatriz(int** X,int filas, int columnas){//la llena de ceros
	for(int i=0;i<filas;i++){
		for(int j=0;j<columnas;j++){
			X[i][j]=1;
		}
	}

}

int main(void){
	int **A,**B,**C,**d_A,**d_B,**d_C,x=2000,y=1500;
	A=(int**)malloc(x*sizeof(int*));//reservamos memoria
	for(int i=0;i<x;i++){
		A[i]=(int*)malloc(y*sizeof(int*));
	}
	B=(int**)malloc(x*sizeof(int*));//reservamos memoria
	for(int i=0;i<x;i++){
		B[i]=(int*)malloc(y*sizeof(int*));
	}
	C=(int**)malloc(x*sizeof(int*));//reservamos memoria
	for(int i=0;i<x;i++){
		C[i]=(int*)malloc(y*sizeof(int*));
	}

	hipMalloc(&d_A,x*sizeof(int));
	for(int i=0;i<x;i++){
		hipMalloc(&d_A[i],y*sizeof(int));
	}
	hipMalloc(&d_B,x*sizeof(int));
	for(int i=0;i<x;i++){
		hipMalloc(&d_B[i],y*sizeof(int));
	}
	hipMalloc(&d_C,x*sizeof(int));
	for(int i=0;i<x;i++){
		hipMalloc(&d_C[i],y*sizeof(int));
	}
	
	inicializaMatriz(A,x,y);
	inicializaMatriz(B,x,y);

	hipMemcpy(&d_A,A,x*y*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(&d_B,B,x*y*sizeof(int),hipMemcpyHostToDevice);
	
	//47*63*1024=3032064  esta es la cantidad de hilos que vamos a utilizar para hacer la suma de las matrices
	//porque las matrices tienen una dimensión de 2000*1500=3000000 
	//32*32 = 1024 hilos en cada bloque
	//2000/32=63, 1500/32=47
	dim3 dimblock(32,32,1);//dimensión de los bloques(cantidad de hilos que se van a utilizar)
	dim3 dimGrid(ceil(x/32),ceil(y/32),1);//dimensión de la malla (cantidad de bloques que se van a utilizar)
	
	SumaMatricesCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,x,y);//enviamos x y y porque son las dimensiones de la matriz y son menos 
	//de los hilos que se van a utilizar

	hipDeviceSynchronize();//espera que termine la funcion anterior 
	hipMemcpy(C,d_C,x*y*sizeof(int),hipMemcpyDeviceToHost);//copia la operacion relizada en el device al host en el vector C

	imprimeMatriz(C,x,y);
	
	free(*A);free(*B);free(*C);free(A);free(B);free(C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
	
	
	return 0;

}

