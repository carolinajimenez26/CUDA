//PROGRAMA QUE SUMA DOS MATRICES EN CUDA
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 


__global__ void SumaMatricesCU(int* A,int* B,int* C,int m,int n){//matriz[m][n]
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int i=row*n+col; //filas*ancho+columnas, el ancho son las columnas que tiene la matriz (eje x)
	if((row<m)&&(col<n)){
		C[i]=A[i]+B[i];
	}	
}

void imprime(int* A,int x, int y){
	for(int i = 0; i < x; i++){
        for(int j = 0; j < y; j++){
            cout<<A[(i*x)+j];
        }
        cout<<endl;
    }
}	

int main(void){
	clock_t start, end;
	int x=2048,y=2048,SIZE=x*y;//dimensiones de la matriz	x = columnas, y = filas
	int *d_A,*d_B,*d_C,*A,*B,*C;

	//reservamos memoria para el host
	A=(int*)malloc(SIZE*sizeof(int));//matriz cuadrada
	B=(int*)malloc(SIZE*sizeof(int));
	C=(int*)malloc(SIZE*sizeof(int));

	//inicializa las matrices A y B
	for(int i=0;i<SIZE;i++){
			A[i]=1;
			B[i]=1;
	}

	//iniciamos la cuenta del reloj
	start = clock();

	//reservamos memoria para el device
	hipMalloc((void**)&d_A,SIZE*sizeof(int));
	hipMalloc((void**)&d_B,SIZE*sizeof(int));
	hipMalloc((void**)&d_C,SIZE*sizeof(int));
	
	//copiamos del host al device
	hipMemcpy(d_A,A,SIZE*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(d_B,B,SIZE*sizeof(int),hipMemcpyHostToDevice);

	dim3 dimblock(32,32,1);//dimensión de los bloques(cantidad de hilos que se van a utilizar)
	dim3 dimGrid(ceil(x/32),ceil(y/32),1);//dimensión de la malla (cantidad de bloques que se van a utilizar)
	
	SumaMatricesCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,y,x);//enviamos x y y porque son las dimensiones de la matriz y son menos 
	//de los hilos que se van a utilizar

	hipDeviceSynchronize();

	hipMemcpy(C,d_C,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	//terminamos la cuenta del reloj
	end = clock();

	imprime(C,x,y);

	cout<<endl;
	cout<<"El tiempo transcurrido fue: "<<((double)(end-start))/CLOCKS_PER_SEC<<endl;

	free(A);free(B);free(C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
	
	return 0;
}

