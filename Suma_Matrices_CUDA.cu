//PROGRAMA QUE SUMA DOS MATRICES EN CUDA
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 


__global__ void SumaMatricesCU(int* A,int* B,int* C,int m,int n){//matriz[m][n]
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int i=row*m+col;
	if((row<m)&&(col<n)){
		C[i]=A[i]+B[i];
	}	
}

void imprime(int* A,int x, int y){
	for(int i = 0; i < x; i++){
        for(int j = 0; j < y; j++){
            cout<<A[(i*x)+j];
        }
        cout<<endl;
    }
}	

int main(void){
	int x=2048,y=2048,SIZE=x*y;//dimensiones de la matriz	x = columnas, y = filas
	int *d_A,*d_B,*d_C,*A,*B,*C;

	//reservamos memoria
	A=(int*)malloc(SIZE*sizeof(int));//matriz cuadrada
	B=(int*)malloc(SIZE*sizeof(int));
	C=(int*)malloc(SIZE*sizeof(int));
	hipMalloc((void**)&d_A,SIZE*sizeof(int));
	hipMalloc((void**)&d_B,SIZE*sizeof(int));
	hipMalloc((void**)&d_C,SIZE*sizeof(int));
	
	//inicializa las matrices A y B
	for(int i=0;i<SIZE;i++){
			A[i]=1;
			B[i]=1;
	}

	hipMemcpy(d_A,A,SIZE*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(d_B,B,SIZE*sizeof(int),hipMemcpyHostToDevice);

	dim3 dimblock(32,32,1);//dimensión de los bloques(cantidad de hilos que se van a utilizar)
	dim3 dimGrid(ceil(x/32),ceil(y/32),1);//dimensión de la malla (cantidad de bloques que se van a utilizar)
	
	SumaMatricesCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,y,x);//enviamos x y y porque son las dimensiones de la matriz y son menos 
	//de los hilos que se van a utilizar

	hipDeviceSynchronize();

	hipMemcpy(C,d_C,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	imprime(C,x,y);

	cout<<endl;
	
	free(A);free(B);free(C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
	

	return 0;

}

