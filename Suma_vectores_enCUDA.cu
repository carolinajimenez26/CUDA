//SUMA DE VECTORES EN C+++
#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>

using namespace std;

//#define SIZE 1024;//tamaño de los vectores
int SIZE=1024;

void inicializaVec(int* X){
	srand(time(NULL));
	for(int i=0;i<SIZE;i++){
                X[i]=rand()%10;
        }
} 

void imprimeVec(int* X){
        for(int i=0;i<SIZE;i++){
                cout<<X[i]<<" ";
        }
}

__global__ void SumaVecCU(int *A,int *B, int*C){
	int tid=threadIdx.x;
	if(tid<SIZE)
		C[tid]=A[tid]+B[tid];

}


int main(void){
	clock_t start = clock();  
	int *A, *B, *C, *d_A, *d_B, *d_C; //vectores a los cuales se le van a realizar las operaciones
	A=(int*)malloc(SIZE*sizeof(int)); 
	B=(int*)malloc(SIZE*sizeof(int));
	C=(int*)malloc(SIZE*sizeof(int));
	hipMalloc(&d_A,SIZE*sizeof(int));
	hipMalloc(&d_B,SIZE*sizeof(int));
	hipMalloc(&d_C,SIZE*sizeof(int));

	hipMemcpy(d_A,A,SIZE*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(d_B,B,SIZE*sizeof(int),hipMemcpyHostToDevice);

	inicializaVec(A);
	inicializaVec(B);

	dim3 dimblock(SIZE,1,1);//vamos a utilicar un bloque con size threads
	dim3 dimGrid(1,1,1);
	
	SumaVecCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C);
	hipDeviceSynchronize();//espera que termine la funcion anterior 
	hipMemcpy(C,d_C,SIZE*sizeof(int),hipMemcpyDeviceToHost);//copia la operacion relizada en el device al host en el vector C

	imprimeVec(C);
	
	cout<<endl<<"Tiempo transcurrido: "<<((double)clock() - start) / CLOCKS_PER_SEC<<endl;
	return 0;
}
