#include "hip/hip_runtime.h"
//SUMA DE VECTORES EN CUDA Y C++ CON TIEMPO
#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>

using namespace std;

#define SIZE 1024;//tamaño de los vectores


__host__ void inicializaVec(int* X){
	srand(time(NULL));
	for(int i=0;i<SIZE;i++){
                X[i]=rand()%10;
        }
} 

__host__ void imprimeVec(int* X){
        for(int i=0;i<SIZE;i++){
                cout<<X[i]<<" ";
        }
}

__host__ void SumaVec(int* X,int* Y,int* Z){
	for(int i=0;i<SIZE;i++){
                Z[i]=X[i]+Y[i];
        }

}

__global__ void SumaVecCU(int *A,int *B, int*C){
	int tid=threadIdx.x;
	//int tid=blockIdx.x;
	if(tid<SIZE)
		C[tid]=A[tid]+B[tid];

}


int main(void){
	clock_t startCPU,endCPU,startGPU,endGPU;  
	int *A, *B, *C, *d_A, *d_B, *d_C,*h_C; //vectores a los cuales se le van a realizar las operaciones

	//-------------------------------CPU--------------------------------------------------------------------
	//iniciamos la cuenta del reloj
	startCPU = clock();	

	//Reservamos memoria para el host
	A=(int*)malloc(SIZE*sizeof(int)); 
	B=(int*)malloc(SIZE*sizeof(int));
	C=(int*)malloc(SIZE*sizeof(int));

	inicializaVec(B);
	inicializaVec(A);
	
	SumaVec(A,B,C);

	//terminamos la cuenta del reloj
	endCPU = clock();
	
	imprimeVec(C);
	double time_CPU=((double)(endCPU-startCPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la GPU fue: "<<time_CPU<<endl;
	//-------------------------------GPU--------------------------------------------------------------------
	h_C=(int*)malloc(SIZE*sizeof(int));
	
	//iniciamos la cuenta del reloj
	startGPU = clock();

	//Reservamos memoria para el device
	hipMalloc(&d_A,SIZE*sizeof(int));
	hipMalloc(&d_B,SIZE*sizeof(int));
	hipMalloc(&d_C,SIZE*sizeof(int));

	hipMemcpy(&d_A,A,SIZE*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(&d_B,B,SIZE*sizeof(int),hipMemcpyHostToDevice);
	
	dim3 dimblock(SIZE,1,1);//vamos a utilicar un bloque con size threads
	dim3 dimGrid(1,1,1);
	
	SumaVecCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C);
	hipDeviceSynchronize();//espera que termine la funcion anterior 
	hipMemcpy(h_C,d_C,SIZE*sizeof(int),hipMemcpyDeviceToHost);//copia la operacion relizada en el device al host en el vector C

	//terminamos la cuenta del reloj	
	endGPU = clock();
	
	imprimeVec(h_C);
	double time_GPU=((double)(endGPU-startGPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la GPU fue: "<<time_GPU<<endl;
	//------------------------------------------------------------------------------------------------------	
	free(A);free(B);free(C);free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	cout<<"El tiempo de aceleramiento fue: "<<time_CPU/time_GPU<<endl;
	return 0;
}
