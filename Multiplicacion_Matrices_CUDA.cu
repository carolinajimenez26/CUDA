#include "hip/hip_runtime.h"
//PROGRAMA QUE MULTIPLICA DOS MATRICES CUADRADAS EN CUDA
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 


__global__ void MultiplicaMatricesCU(int* A,int* B,int* C,int m,int n){//matriz[m][n]
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int i=row*n+col;
	// ancho=n (columnas, o sea en x)
	if((row<m)&&(col<n)){
		int suma=0;
		for(int k=0;k<m;k++){//este es para que multiplique por las columnas 
			suma=suma+A[(row*n)+k]*B[(row*n)+k];
		}
		C[i]=suma;
	}	
}

void imprime(int* A,int filas, int columnas){//imprime como si fuera una matriz
	for(int i = 0; i < x; i++){
        	for(int j = 0; j < y; j++){
            		cout<<A[(i*x)+j];
        	}
        cout<<endl;
    }
}	

void inicializa(int *A,int filas, int columnas){//inicializa arreglos
	for(int i=0;i<x*y;i++){
		A[i]=1;
	}
}

int main(void){
	clock_t start, end;//para contar el tiempo de ejecución del programa
	int x=2048,y=2048,SIZE=x*y;//dimensiones de la matriz	x = columnas, y = filas
	int *d_A,*d_B,*d_C,*A,*B,*C;
	
	//reservamos memoria en el host
	A=(int*)malloc(SIZE*sizeof(int));//matriz cuadrada
	B=(int*)malloc(SIZE*sizeof(int));
	C=(int*)malloc(SIZE*sizeof(int));

	//inicializa las matrices A y B
	inicializa(A,y,x);
	inicializa(B,y,x);

	//iniciamos la cuenta del reloj
	start = clock();

	//reservamos memoria en el device
	hipMalloc((void**)&d_A,SIZE*sizeof(int));
	hipMalloc((void**)&d_B,SIZE*sizeof(int));
	hipMalloc((void**)&d_C,SIZE*sizeof(int));
	
	//copiamos del host al device
	hipMemcpy(d_A,A,SIZE*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(d_B,B,SIZE*sizeof(int),hipMemcpyHostToDevice);

	dim3 dimblock(32,32,1);//dimensión de los bloques(cantidad de hilos que se van a utilizar)
	dim3 dimGrid(ceil(x/32),ceil(y/32),1);//dimensión de la malla (cantidad de bloques que se van a utilizar)
	
	MultiplicaMatricesCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,y,x);//enviamos x y y porque son las dimensiones de la matriz y se deben enviar para no utilizar más hilos o bloques de los necesarios

	hipDeviceSynchronize();//esperamos a que termine la función anterior

	hipMemcpy(C,d_C,SIZE*sizeof(int),hipMemcpyDeviceToHost);

	//terminamos la cuenta del reloj
	end = clock();

	imprime(C,y,x);//mostramos el resultado

	cout<<endl;

	cout<<"El tiempo transcurrido fue: "<<((double)(end-start))/CLOCKS_PER_SEC<<endl;	

	//liberamos memoria tanto la utilizada en el host como la del device
	free(A);free(B);free(C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
	
	return 0;
}

