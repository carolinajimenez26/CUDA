//SUMA DE MATRICES EN C++ Y CUDA CON TIEMPO
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 

__global__ void SumaCU(int* A,int* B,int* C,int m,int n){//matriz[m][n]
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if((row<m)&&(col<n)){
		C[row*n+col]=A[row*n+col]+B[row*n+col];
	}	
}

__host__ void suma(int* A,int* B,int* C,int filas,int columnas){//Para sumar dos matrices deben tener las mismas dimensiones
	for(int i = 0; i < filas ; i++){
        	for(int j = 0; j < columnas ; j++){
                	C[(i*columnas)+j]=A[(i*columnas)+j]+B[(i*columnas)+j];
        	}
   	}
}

__host__ void imprime(int* A,int filas, int columnas){//imprime como si fuera una matriz
	for(int i = 0; i < filas; i++){
        	for(int j = 0; j < columnas; j++){
            		cout<<A[(i*columnas)+j];
        	}
        cout<<endl;
    }
}	

__host__ void inicializa(int *A,int filas, int columnas){//inicializa arreglos
	for(int i=0;i<filas*columnas;i++){
		A[i]=1;
	}
}

int main(void){

	clock_t startCPU,endCPU,startGPU,endGPU;  
	int *A,*B,*C,*h_C,*d_A,*d_B,*d_C;
	int filas=2048,columnas=2048,SIZE=filas*columnas*sizeof(int);

	//-------------------------------CPU--------------------------------------------------------------------
	startCPU = clock();	

	A=(int*)malloc(SIZE); 
	B=(int*)malloc(SIZE);
	C=(int*)malloc(SIZE);

	inicializa(A,filas,columnas);
	inicializa(B,filas,columnas);	

	suma(A,B,C,filas,columnas);
	
	endCPU = clock();

	//imprime(C,filas,columnas);
	double time_CPU=((double)(endCPU-startCPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la CPU fue: "<<time_CPU<<endl;
	//-------------------------------GPU--------------------------------------------------------------------	
	h_C=(int*)malloc(SIZE);
	
	startGPU = clock();

	hipMalloc((void**)&d_A,SIZE);
	hipMalloc((void**)&d_B,SIZE);
	hipMalloc((void**)&d_C,SIZE);

	hipMemcpy(d_A,A,SIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,SIZE,hipMemcpyHostToDevice);

	dim3 dimblock(32,32,1);//ya que es una matriz 10x10
	dim3 dimGrid(ceil(filas/32),ceil(columnas/32),1);
	
	SumaCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,filas,columnas);
	hipDeviceSynchronize();//espera que termine la funcion anterior 
	hipMemcpy(h_C,d_C,SIZE,hipMemcpyDeviceToHost);//copia la operacion relizada en el device al host en el vector C
	
	endGPU = clock();
	
	imprime(h_C,filas,columnas);
	double time_GPU=((double)(endGPU-startGPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la GPU fue: "<<time_GPU<<endl;
	//------------------------------------------------------------------------------------------------------	
	free(A);free(B);free(C);free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	cout<<"El tiempo de aceleramiento fue: "<<time_CPU/time_GPU<<endl;
	
	return 0;
}

